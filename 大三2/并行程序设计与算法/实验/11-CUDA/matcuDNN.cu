#include <stdio.h>
#include <sys/time.h>
#include <fstream>
#include <vector>
#include <hipDNN.h>

double cpuSecond()//获取当前时间,转化为微秒单位
{
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return((double)tp.tv_sec*1e6+(double)tp.tv_usec);
}

void show(float* A, int n, int show_n){// 展示左上角 n*n 部分矩阵
    for (int i = 0; i < show_n; i++) {
        for (int j = 0; j < show_n; j++) {
            printf("%12.6f", A[i*n + j]);
        }
        printf("\n");
    }
}

void loadMat(const char* filename, float* tensor, int size) { // 导入输入二进制文件input和kernel
    FILE* file = fopen(filename, "rb");
    fread(tensor, sizeof(float), size, file);
    fclose(file);
}



int main() {

    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    double iStart,iElaps;    // 时间记录
    hipdnnCreate(&cudnn);

    const int batch_size = 1;
    const int in_channels = 3;
    const int in_height = 256;
    const int in_width = 256;
    const int kernel_height = 3;
    const int kernel_width = 3;
    const int out_channels = 1;
    //const int padding = 1;
    //const int stride = 1;
    int stride[3] = {1,2,3};
    int padding[3] = {0,1,1};
    int out_height[3],out_width[3],output_size[3];
    int input_size = in_height * in_width * in_channels;
    int kernel_size = in_channels * kernel_height * kernel_width;
    for(int i=0;i<3;i++){
        out_width[i] = (in_width + 2 * padding[i] - 3) / stride[i] + 1;
        out_height[i] = (in_height + 2 * padding[i] - 3) / stride[i] + 1;
        output_size[i] = out_width[i] * out_height[i];
    }

    // input 
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnCreateTensorDescriptor(&input_descriptor);   
    hipdnnSetTensor4dDescriptor(input_descriptor,HIPDNN_TENSOR_NHWC,HIPDNN_DATA_FLOAT,
                                    batch_size,in_channels,in_height,in_width);
    // kernel
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    hipdnnSetFilter4dDescriptor(kernel_descriptor,HIPDNN_DATA_FLOAT,HIPDNN_TENSOR_NCHW,
                                   out_channels,in_channels,kernel_height,kernel_width);
                                   
    // output_1
    hipdnnTensorDescriptor_t output_descriptor_1;
    hipdnnCreateTensorDescriptor(&output_descriptor_1);
    hipdnnSetTensor4dDescriptor(output_descriptor_1,HIPDNN_TENSOR_NHWC,HIPDNN_DATA_FLOAT,
                                    batch_size,out_channels,out_height[0],out_width[0]);
    // output_2
    hipdnnTensorDescriptor_t output_descriptor_2;
    hipdnnCreateTensorDescriptor(&output_descriptor_2);
    hipdnnSetTensor4dDescriptor(output_descriptor_2,HIPDNN_TENSOR_NHWC,HIPDNN_DATA_FLOAT,
                                    batch_size,out_channels,out_height[1],out_width[1]);
    // output_3
    hipdnnTensorDescriptor_t output_descriptor_3;
    hipdnnCreateTensorDescriptor(&output_descriptor_3);
    hipdnnSetTensor4dDescriptor(output_descriptor_3,HIPDNN_TENSOR_NHWC,HIPDNN_DATA_FLOAT,
                                    batch_size,out_channels,out_height[2],out_width[2]);         
    // convolution_1
    hipdnnConvolutionDescriptor_t convolution_descriptor_1;
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor_1);
    hipdnnSetConvolution2dDescriptor(convolution_descriptor_1,
                                    padding[0], padding[0],
                                    stride[0], stride[0],
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION,HIPDNN_DATA_FLOAT);
    // convolution_2
    hipdnnConvolutionDescriptor_t convolution_descriptor_2;
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor_2);
    hipdnnSetConvolution2dDescriptor(convolution_descriptor_2,
                                    padding[1], padding[1],
                                    stride[1], stride[1],
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION,HIPDNN_DATA_FLOAT);
    // convolution_3
    hipdnnConvolutionDescriptor_t convolution_descriptor_3;
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor_3);
    hipdnnSetConvolution2dDescriptor(convolution_descriptor_3,
                                    padding[2], padding[2],
                                    stride[2], stride[2],
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION,HIPDNN_DATA_FLOAT);
    // convolution algorithm_1
    hipdnnConvolutionFwdAlgo_t convolution_algorithm_1;
    hipdnnGetConvolutionForwardAlgorithm(cudnn,input_descriptor,kernel_descriptor,convolution_descriptor_1,
                        output_descriptor_1,HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,0,&convolution_algorithm_1);
    // convolution algorithm_2
    hipdnnConvolutionFwdAlgo_t convolution_algorithm_2;
    hipdnnGetConvolutionForwardAlgorithm(cudnn,input_descriptor,kernel_descriptor,convolution_descriptor_2,
                        output_descriptor_2,HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,0,&convolution_algorithm_2);
    // convolution algorithm_3
    hipdnnConvolutionFwdAlgo_t convolution_algorithm_3;
    hipdnnGetConvolutionForwardAlgorithm(cudnn,input_descriptor,kernel_descriptor,convolution_descriptor_3,
                        output_descriptor_3,HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,0,&convolution_algorithm_3);
    
    void *d_workspace_1, *d_workspace_2, *d_workspace_3;
    // Get workspace size_1
    size_t workspace_bytes_1 = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn,input_descriptor,kernel_descriptor,convolution_descriptor_1,
                                                output_descriptor_1,convolution_algorithm_1,&workspace_bytes_1);
    hipMalloc(&d_workspace_1, workspace_bytes_1);
    // Get workspace size_2
    size_t workspace_bytes_2 = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn,input_descriptor,kernel_descriptor,convolution_descriptor_2,
                                                output_descriptor_2,convolution_algorithm_2,&workspace_bytes_2);
    hipMalloc(&d_workspace_2, workspace_bytes_2);
    // Get workspace size_3
    size_t workspace_bytes_3 = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn,input_descriptor,kernel_descriptor,convolution_descriptor_3,
                                                output_descriptor_3,convolution_algorithm_3,&workspace_bytes_3);
    hipMalloc(&d_workspace_3, workspace_bytes_3);

    // Allocate memory for input, kernel, and output
    float *d_input, *d_kernel,*d_output_1,*d_output_2,*d_output_3;
    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_kernel, kernel_size * sizeof(float));
    hipMalloc(&d_output_1, output_size[0] * sizeof(float));
    hipMalloc(&d_output_2, output_size[1] * sizeof(float));
    hipMalloc(&d_output_3, output_size[2] * sizeof(float));
    /////////////////到这
    // Initialize input and kernel
    float* h_input = (float*)malloc(input_size * sizeof(float));
    float* h_kernel_1 = (float*)malloc(kernel_size * sizeof(float));
    float* h_kernel_2 = (float*)malloc(kernel_size * sizeof(float));
    float* h_kernel_3 = (float*)malloc(kernel_size * sizeof(float));
    float* h_output_1 = (float*)malloc(output_size[0] * sizeof(float));
    float* h_output_2 = (float*)malloc(output_size[1] * sizeof(float));
    float* h_output_3 = (float*)malloc(output_size[2] * sizeof(float));
    loadMat("input_tensor.bin", h_input, input_size);
    loadMat("kernel_tensor_1.bin", h_kernel_1, kernel_size);
    loadMat("kernel_tensor_2.bin", h_kernel_2, kernel_size);
    loadMat("kernel_tensor_3.bin", h_kernel_3, kernel_size);
    
    // hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel_1, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    // Perform convolution
    const float alpha = 1.0f;
    const float beta = 0.0f;
    // 开始时间
    iStart=cpuSecond();

    hipdnnConvolutionForward(cudnn,&alpha,
                                input_descriptor,d_input,
                                kernel_descriptor,d_kernel,
                                convolution_descriptor_1,convolution_algorithm_1,
                                d_workspace_1,workspace_bytes_1,
                                &beta,output_descriptor_1,d_output_1);
    hipDeviceSynchronize();
    hipMemcpy(h_output_1, d_output_1, output_size[0] * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_kernel, h_kernel_2, kernel_size * sizeof(float), hipMemcpyHostToDevice);
    hipdnnConvolutionForward(cudnn,&alpha,
                                input_descriptor,d_input,
                                kernel_descriptor,d_kernel,
                                convolution_descriptor_2,convolution_algorithm_2,
                                d_workspace_2,workspace_bytes_2,
                                &beta,output_descriptor_2,d_output_2);
    hipDeviceSynchronize();
    hipMemcpy(h_output_2, d_output_2, output_size[1] * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_kernel, h_kernel_3, kernel_size * sizeof(float), hipMemcpyHostToDevice);
    hipdnnConvolutionForward(cudnn,&alpha,
                                input_descriptor,d_input,
                                kernel_descriptor,d_kernel,
                                convolution_descriptor_3,convolution_algorithm_3,
                                d_workspace_3,workspace_bytes_3,
                                &beta,output_descriptor_3,d_output_3);
    hipDeviceSynchronize();
    hipMemcpy(h_output_3, d_output_3, output_size[2] * sizeof(float), hipMemcpyDeviceToHost);
    // 结束时间
    iElaps=cpuSecond()-iStart;
    printf("Time taken: %f us\n",iElaps);
    printf(" Top left corner of Conv output_1: \n");
    show(h_output_1,out_width[0],6);
    printf(" Top left corner of Conv output_2: \n");
    show(h_output_2,out_width[1],6);
    printf(" Top left corner of Conv output_3: \n");
    show(h_output_3,out_width[2],6);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output_1);
    hipFree(d_output_2);
    hipFree(d_output_3);
    hipFree(d_workspace_1);
    hipFree(d_workspace_2);
    hipFree(d_workspace_3);
    
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor_1);
    hipdnnDestroyTensorDescriptor(output_descriptor_2);
    hipdnnDestroyTensorDescriptor(output_descriptor_3);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor_1);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor_2);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor_3);
    hipdnnDestroy(cudnn);
    
    free(h_input);
    free(h_kernel_1);
    free(h_kernel_2);
    free(h_kernel_3);
    free(h_output_1);
    free(h_output_2);
    free(h_output_3);
    return 0;
}
